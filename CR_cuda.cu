#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

__global__ void init_f(double *f, double delta)
{
    int i = threadIdx.x;
    f[i]=sin(5*i*delta);
}

__global__ void init_abcd(double *a, double *b, double *c, double *d, double *f, double delta){
    int i = threadIdx.x;
    if(i != 0){
        a[i] = 1.0;
        b[i] = 4.0;
        c[i] = 1.0;
        d[i] = 3*(f[i+1]-f[i-1])/delta;
    }
}

__global__ void equate(double *a, double *b, double *c, double *d, double *a1, double *b1, double *c1, double *d1){
    int i = threadIdx.x;
    a1[i]=a[i];
    b1[i]=b[i];
    c1[i]=c[i];
    d1[i]=d[i];
}

__global__ void cyclic_reduction_loop(double *a, double *b, double *c, double *d, double *a1, double *b1, double *c1, double *d1, int n, int k){
    int i = (pow(2,k)-1) + threadIdx.x*pow(2,k);
    double alpha_i,beta_i;
    int up,dwn;
    up = i-pow(2,k-1);
    dwn= i+pow(2,k-1);
    alpha_i= (-1)*a[i]/b[up];
    beta_i= (-1)*c[i]/b[dwn];
    //printf("%d , %d , %d\n",i,up,dwn);
    if(i-pow(2,k)<0)
    {
        a1[i]=0;
    }
    else
    {
        a1[i]=a[up]*alpha_i;
    }
    if(dwn>=n)
    {
        b1[i]=b[i]+c[up]*alpha_i;
    }
    else{
        b1[i]=b[i]+a[dwn]*beta_i+c[up]*alpha_i;
    }

    if(dwn>=n)
    {
        c1[i]=0;
    }
    else
    {
        c1[i]=c[dwn]*beta_i;
    }
    if(dwn>=n)
    {
        d1[i]=d[i]+alpha_i*d[up];
    }
    else{
        d1[i]=d[i]+alpha_i*d[up]+beta_i*d[dwn];
    }
}

__global__ void cyclic_reduction_loop(double *a, double *b, double *c, double *d, int n, int k){
    int i = (pow(2,k)-1) + threadIdx.x*pow(2,k+1);
    double sub_alpha, sub_beta;
    int up,dwn;
    up=i-pow(2,k);
    dwn=i+pow(2,k);
    if(up<0)
    {
        sub_beta=-c[i]/b[dwn];
        d1[i]=d[i]+d[dwn]*sub_beta;
    }
    else
    {
        if(dwn>n-1)
        {
            sub_alpha=-a[i]/b[up];
            d1[i]=d[i]+d[up]*sub_alpha;
        }
        else
        {
            sub_beta=-c[i]/b[dwn];
            sub_alpha=-a[i]/b[up];
            d1[i]=d[i]+d[up]*sub_alpha+d[dwn]*sub_beta;
        }
    }
}

__global__ void backprop(double *b, double *d){
    int i = threadIdx.x;
    d[i]/=b[i];
}

int main()
{
    clock_t start=clock();
    int n=1001;
    hipMalloc((void**)&a, n*sizeof(double));
    hipMalloc((void**)&b, n*sizeof(double));
    hipMalloc((void**)&c, n*sizeof(double));
    hipMalloc((void**)&d, n*sizeof(double));
    hipMalloc((void**)&f, n*sizeof(double));
    double delta=0.003;

    init_f<<< 1,n >>>(f, delta);
    init_abcd<<< 1,n-1 >>>(a,b,c,d,f,delta);

    a[0]=0;
    b[0]=1;
    c[0]=2;
    c[n-1]=0;
    a[n-1]=2;
    b[n-1]=1;
    d[n-1]=(2.5*f[n-1]-2*f[n-2]-0.5*f[n-3])/delta;
    d[0]=(-2.5*f[0]+2*f[1]+0.5*f[2])/delta;
    
    hipMalloc((void**)&a1, n*sizeof(double));
    hipMalloc((void**)&b1, n*sizeof(double));
    hipMalloc((void**)&c1, n*sizeof(double));
    hipMalloc((void**)&d1, n*sizeof(double));
    int logn;
    logn = log(n)/log(2);
    //// cyclic reduction 
    for(int k=1;k<=logn;k++)
    {
        equate<<< 1,n >>>(a,b,c,d,a1,b1,c1,d1);
        int n_threads = (n - (pow(2,k)-1))/pow(2,k);
        cyclic_reduction_loop<<< 1,n_threads >>>(a,b,c,d,a1,b1,c1,d1,n,k);
        equate<<< 1,n >>>(a1,b1,c1,d1,a,b,c,d);

    }
    for(int k=logn-1; k>=0;k--)
    {
        equate<<< 1,n >>>(a,b,c,d,a1,b1,c1,d1);
        int n_threads = (n - (pow(2,k)-1))/pow(2,k+1);
        cyclic_reduction_loop<<< 1,n_threads >>>(a,b,c,d,n,k);
        equate<<< 1,n >>>(a1,b1,c1,d1,a,b,c,d);
    }
    backprop<<< 1,n >>>(b,d);

    double d_host[n];
    hipMemcpy(d_host, d, n*sizeof(double), hipMemcpyDeviceToHost);
    for(int i=0;i<n;i++)
    {
        printf("%lf ,", d_host[i]);
    }
    clock_t end=clock();
    printf("Time: %ld\n",end-start);
    return 0;
}
